#include "hip/hip_runtime.h"
#include <iostream>
#include <cppsim/state_gpu.hpp>
#include <cppsim/state.hpp>
#include <cppsim/gate_factory.hpp>
#include <cppsim/gate_matrix.hpp>
#include <cppsim/gate.hpp>
#include <cppsim/utility.hpp>
#include <cppsim/pauli_operator.hpp>
#include <time.h>
#include <vector>

using namespace std;

void test(int, int, vector<double>&);
void dbg(vector<double>);

int main(){
    int qubit_start = 4;
    int qubit_end = 20;
    int repeat = 100;
    cin >> qubit_end >> repeat;
    qubit_end = qubit_end >= qubit_start ? qubit_end : qubit_start;

    clock_t start,end;
    vector<double> time_list;

    for(int i=qubit_start;i<=qubit_end;i++){
        test(i, repeat, time_list);
    }
    dbg(time_list);
}

void test(int qubit_num, int repeat, vector<double>& time_list){
    clock_t start,end;
    start = clock();
    for(int i=0;i<repeat;i++){
        QuantumStateGpu state(qubit_num);
        state.set_Haar_random_state();
        auto gate = gate::X(0);
        gate->update_quantum_state(&state);
    }
    end = clock();
    time_list.push_back((double)(end-start)/CLOCKS_PER_SEC/repeat);
}

void dbg(vector<double> time_list){
    for(int i=0;i<time_list.size();i++){
        cout << scientific << setprecision(1) << time_list[i] << " ";
    }
    cout << endl;
}

